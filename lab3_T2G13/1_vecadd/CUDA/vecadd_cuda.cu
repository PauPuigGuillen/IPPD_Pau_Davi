
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define BLOCKSIZE 128

// CUDA kernel for vector addition with the required signature
__global__ void vecadd_cuda(double *A, double *B, double *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        C[i] = A[i] + B[i];
    }
}

// CUDA vector addition wrapper function with detailed timing
void vecadd_wrapper(double *h_A, double *h_B, double *h_C, const int N)
{
    double *d_A, *d_B, *d_C;
    size_t size = N * sizeof(double);
    float h2d_time = 0.0f, kernel_time = 0.0f, d2h_time = 0.0f;
    
    // Event for timing host to device, kernel execution, and device to host (in that order)
    hipEvent_t start_h2d, end_h2d;
    hipEvent_t start_kernel, end_kernel;
    hipEvent_t start_d2h, end_d2h;
    
    hipEventCreate(&start_h2d);
    hipEventCreate(&end_h2d);
    hipEventCreate(&start_kernel);
    hipEventCreate(&end_kernel);
    hipEventCreate(&start_d2h);
    hipEventCreate(&end_d2h);

    // Allocate device memory
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Time Host to Device copy
    hipMemset(d_C, 0, size);
    hipEventRecord(start_h2d, 0);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    hipEventRecord(end_h2d, 0);
    hipEventSynchronize(end_h2d);
    hipEventElapsedTime(&h2d_time, start_h2d, end_h2d);

    // Time kernel execution
    hipEventRecord(start_kernel, 0);
    int numBlocks = (N + BLOCKSIZE - 1) / BLOCKSIZE;
    vecadd_cuda<<<numBlocks, BLOCKSIZE>>>(d_A, d_B, d_C, N);
    //cudaDeviceSynchronize();
    hipEventRecord(end_kernel, 0);
    hipEventSynchronize(end_kernel);
    hipEventElapsedTime(&kernel_time, start_kernel, end_kernel);

    // Time Device to Host copy
    hipEventRecord(start_d2h, 0);
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    hipEventRecord(end_d2h, 0);
    hipEventSynchronize(end_d2h);
    hipEventElapsedTime(&d2h_time, start_d2h, end_d2h);

    // Print timing information
    printf(" Copy A and B Host to Device elapsed time: %.9f seconds\n", h2d_time / 1000.0f);
    printf(" Kernel elapsed time: %.9f seconds\n", kernel_time / 1000.0f);
    printf(" Copy C Device to Host elapsed time: %.9f seconds\n", d2h_time / 1000.0f);
    printf(" Total elapsed time: %.9f seconds\n", (h2d_time + kernel_time + d2h_time) / 1000.0f);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    // Clean up events
    hipEventDestroy(start_h2d);
    hipEventDestroy(end_h2d);
    hipEventDestroy(start_kernel);
    hipEventDestroy(end_kernel);
    hipEventDestroy(start_d2h);
    hipEventDestroy(end_d2h);
}

int main(int argc, char *argv[])
{
    int N;

    if (argc != 2)
    {
        printf("Usage: %s <vector size N>\n", argv[0]);
        return 1;
    }
    else
    {
        N = atoi(argv[1]);
    }
    printf("Vector size: %d\n", N);

    // Memory allocation
    double *A = (double *)malloc(N * sizeof(double));
    double *B = (double *)malloc(N * sizeof(double));
    double *C = (double *)malloc(N * sizeof(double));

    if (A == NULL || B == NULL || C == NULL) {
        fprintf(stderr, "Host memory allocation failed\n");
        return 1;
    }

    // Initialize vectors
    for (int i = 0; i < N; i++)
    {
        A[i] = (double)i;
        B[i] = 2.0 * (N - i);
    }

    // Call the wrapper with CUDA event timing
    vecadd_wrapper(A, B, C, N);

    // Validation
    int errors = 0;
    for (int i = 0; i < N; i++)
    {
        if (fabs(C[i] - (2.0 * N - i)) > 1e-6)
        {
            printf("Validation failed at index %d: C[%d] = %f, expected = %f\n", 
                   i, i, C[i], 2.0 * N - i);
            errors++;
        }
    }
    
    if (errors == 0) {
        printf("Validation successful! All values match expected results.\n");
    }

    // Free memory
    free(A);
    free(B);
    free(C);

    return 0;
}